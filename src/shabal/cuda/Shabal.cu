#include "hip/hip_runtime.h"
/* $Id: shabal.c 175 2010-05-07 16:03:20Z tp $ */
/*
* Shabal implementation.
*
* ==========================(LICENSE BEGIN)============================
*
* Copyright (c) 2007-2010  Projet RNRT SAPHIR
*
* Permission is hereby granted, free of charge, to any person obtaining
* a copy of this software and associated documentation files (the
* "Software"), to deal in the Software without restriction, including
* without limitation the rights to use, copy, modify, merge, publish,
* distribute, sublicense, and/or sell copies of the Software, and to
* permit persons to whom the Software is furnished to do so, subject to
* the following conditions:
*
* The above copyright notice and this permission notice shall be
* included in all copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
* EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
* MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
* CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
* TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*
* ===========================(LICENSE END)=============================
*
* @author            Thomas Pornin <thomas.pornin@cryptolog.com>
* @cuda author		 Creepsky      <creepsky@gmail.com>
* 
*/

#include "shabal/sphlib/sph_shabal.h"
#include "Shabal.hpp"
#include <string>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>

using namespace Burst;

#define SPH_C32(x)    ((sph_u32)(x ## U))
#define C32   SPH_C32

#define sM    16

#define C32   SPH_C32
#define T32   SPH_T32

#define O1   13
#define O2    9
#define O3    6

/*
* We copy the state into local variables, so that the compiler knows
* that it can optimize them at will.
*/

/* BEGIN -- automatically generated code. */

#define DECL_STATE   \
	sph_u32 A00, A01, A02, A03, A04, A05, A06, A07, \
	        A08, A09, A0A, A0B; \
	sph_u32 B0, B1, B2, B3, B4, B5, B6, B7, \
	        B8, B9, BA, BB, BC, BD, BE, BF; \
	sph_u32 C0, C1, C2, C3, C4, C5, C6, C7, \
	        C8, C9, CA, CB, CC, CD, CE, CF; \
	sph_u32 M0, M1, M2, M3, M4, M5, M6, M7, \
	        M8, M9, MA, MB, MC, MD, ME, MF; \
	sph_u32 Wlow, Whigh;

#define READ_STATE(state)   do { \
		A00 = (state)->A[0]; \
		A01 = (state)->A[1]; \
		A02 = (state)->A[2]; \
		A03 = (state)->A[3]; \
		A04 = (state)->A[4]; \
		A05 = (state)->A[5]; \
		A06 = (state)->A[6]; \
		A07 = (state)->A[7]; \
		A08 = (state)->A[8]; \
		A09 = (state)->A[9]; \
		A0A = (state)->A[10]; \
		A0B = (state)->A[11]; \
		B0 = (state)->B[0]; \
		B1 = (state)->B[1]; \
		B2 = (state)->B[2]; \
		B3 = (state)->B[3]; \
		B4 = (state)->B[4]; \
		B5 = (state)->B[5]; \
		B6 = (state)->B[6]; \
		B7 = (state)->B[7]; \
		B8 = (state)->B[8]; \
		B9 = (state)->B[9]; \
		BA = (state)->B[10]; \
		BB = (state)->B[11]; \
		BC = (state)->B[12]; \
		BD = (state)->B[13]; \
		BE = (state)->B[14]; \
		BF = (state)->B[15]; \
		C0 = (state)->C[0]; \
		C1 = (state)->C[1]; \
		C2 = (state)->C[2]; \
		C3 = (state)->C[3]; \
		C4 = (state)->C[4]; \
		C5 = (state)->C[5]; \
		C6 = (state)->C[6]; \
		C7 = (state)->C[7]; \
		C8 = (state)->C[8]; \
		C9 = (state)->C[9]; \
		CA = (state)->C[10]; \
		CB = (state)->C[11]; \
		CC = (state)->C[12]; \
		CD = (state)->C[13]; \
		CE = (state)->C[14]; \
		CF = (state)->C[15]; \
		Wlow = (state)->Wlow; \
		Whigh = (state)->Whigh; \
	} while (0)

#define WRITE_STATE(state)   do { \
		(state)->A[0] = A00; \
		(state)->A[1] = A01; \
		(state)->A[2] = A02; \
		(state)->A[3] = A03; \
		(state)->A[4] = A04; \
		(state)->A[5] = A05; \
		(state)->A[6] = A06; \
		(state)->A[7] = A07; \
		(state)->A[8] = A08; \
		(state)->A[9] = A09; \
		(state)->A[10] = A0A; \
		(state)->A[11] = A0B; \
		(state)->B[0] = B0; \
		(state)->B[1] = B1; \
		(state)->B[2] = B2; \
		(state)->B[3] = B3; \
		(state)->B[4] = B4; \
		(state)->B[5] = B5; \
		(state)->B[6] = B6; \
		(state)->B[7] = B7; \
		(state)->B[8] = B8; \
		(state)->B[9] = B9; \
		(state)->B[10] = BA; \
		(state)->B[11] = BB; \
		(state)->B[12] = BC; \
		(state)->B[13] = BD; \
		(state)->B[14] = BE; \
		(state)->B[15] = BF; \
		(state)->C[0] = C0; \
		(state)->C[1] = C1; \
		(state)->C[2] = C2; \
		(state)->C[3] = C3; \
		(state)->C[4] = C4; \
		(state)->C[5] = C5; \
		(state)->C[6] = C6; \
		(state)->C[7] = C7; \
		(state)->C[8] = C8; \
		(state)->C[9] = C9; \
		(state)->C[10] = CA; \
		(state)->C[11] = CB; \
		(state)->C[12] = CC; \
		(state)->C[13] = CD; \
		(state)->C[14] = CE; \
		(state)->C[15] = CF; \
		(state)->Wlow = Wlow; \
		(state)->Whigh = Whigh; \
	} while (0)

#define DECODE_BLOCK   do { \
		M0 = cuda_sph_dec32le_aligned(buf + 0); \
		M1 = cuda_sph_dec32le_aligned(buf + 4); \
		M2 = cuda_sph_dec32le_aligned(buf + 8); \
		M3 = cuda_sph_dec32le_aligned(buf + 12); \
		M4 = cuda_sph_dec32le_aligned(buf + 16); \
		M5 = cuda_sph_dec32le_aligned(buf + 20); \
		M6 = cuda_sph_dec32le_aligned(buf + 24); \
		M7 = cuda_sph_dec32le_aligned(buf + 28); \
		M8 = cuda_sph_dec32le_aligned(buf + 32); \
		M9 = cuda_sph_dec32le_aligned(buf + 36); \
		MA = cuda_sph_dec32le_aligned(buf + 40); \
		MB = cuda_sph_dec32le_aligned(buf + 44); \
		MC = cuda_sph_dec32le_aligned(buf + 48); \
		MD = cuda_sph_dec32le_aligned(buf + 52); \
		ME = cuda_sph_dec32le_aligned(buf + 56); \
		MF = cuda_sph_dec32le_aligned(buf + 60); \
	} while (0)

#define INPUT_BLOCK_ADD   do { \
		B0 = T32(B0 + M0); \
		B1 = T32(B1 + M1); \
		B2 = T32(B2 + M2); \
		B3 = T32(B3 + M3); \
		B4 = T32(B4 + M4); \
		B5 = T32(B5 + M5); \
		B6 = T32(B6 + M6); \
		B7 = T32(B7 + M7); \
		B8 = T32(B8 + M8); \
		B9 = T32(B9 + M9); \
		BA = T32(BA + MA); \
		BB = T32(BB + MB); \
		BC = T32(BC + MC); \
		BD = T32(BD + MD); \
		BE = T32(BE + ME); \
		BF = T32(BF + MF); \
	} while (0)

#define INPUT_BLOCK_SUB   do { \
		C0 = T32(C0 - M0); \
		C1 = T32(C1 - M1); \
		C2 = T32(C2 - M2); \
		C3 = T32(C3 - M3); \
		C4 = T32(C4 - M4); \
		C5 = T32(C5 - M5); \
		C6 = T32(C6 - M6); \
		C7 = T32(C7 - M7); \
		C8 = T32(C8 - M8); \
		C9 = T32(C9 - M9); \
		CA = T32(CA - MA); \
		CB = T32(CB - MB); \
		CC = T32(CC - MC); \
		CD = T32(CD - MD); \
		CE = T32(CE - ME); \
		CF = T32(CF - MF); \
	} while (0)

#define XOR_W   do { \
		A00 ^= Wlow; \
		A01 ^= Whigh; \
	} while (0)

#define SWAP(v1, v2)   do { \
		sph_u32 tmp = (v1); \
		(v1) = (v2); \
		(v2) = tmp; \
	} while (0)

#define SWAP_BC   do { \
		SWAP(B0, C0); \
		SWAP(B1, C1); \
		SWAP(B2, C2); \
		SWAP(B3, C3); \
		SWAP(B4, C4); \
		SWAP(B5, C5); \
		SWAP(B6, C6); \
		SWAP(B7, C7); \
		SWAP(B8, C8); \
		SWAP(B9, C9); \
		SWAP(BA, CA); \
		SWAP(BB, CB); \
		SWAP(BC, CC); \
		SWAP(BD, CD); \
		SWAP(BE, CE); \
		SWAP(BF, CF); \
	} while (0)

#define PERM_ELT(xa0, xa1, xb0, xb1, xb2, xb3, xc, xm)   do { \
		xa0 = T32((xa0 \
			^ (((xa1 << 15) | (xa1 >> 17)) * 5U) \
			^ xc) * 3U) \
			^ xb1 ^ (xb2 & ~xb3) ^ xm; \
		xb0 = T32(~(((xb0 << 1) | (xb0 >> 31)) ^ xa0)); \
	} while (0)

#define PERM_STEP_0   do { \
		PERM_ELT(A00, A0B, B0, BD, B9, B6, C8, M0); \
		PERM_ELT(A01, A00, B1, BE, BA, B7, C7, M1); \
		PERM_ELT(A02, A01, B2, BF, BB, B8, C6, M2); \
		PERM_ELT(A03, A02, B3, B0, BC, B9, C5, M3); \
		PERM_ELT(A04, A03, B4, B1, BD, BA, C4, M4); \
		PERM_ELT(A05, A04, B5, B2, BE, BB, C3, M5); \
		PERM_ELT(A06, A05, B6, B3, BF, BC, C2, M6); \
		PERM_ELT(A07, A06, B7, B4, B0, BD, C1, M7); \
		PERM_ELT(A08, A07, B8, B5, B1, BE, C0, M8); \
		PERM_ELT(A09, A08, B9, B6, B2, BF, CF, M9); \
		PERM_ELT(A0A, A09, BA, B7, B3, B0, CE, MA); \
		PERM_ELT(A0B, A0A, BB, B8, B4, B1, CD, MB); \
		PERM_ELT(A00, A0B, BC, B9, B5, B2, CC, MC); \
		PERM_ELT(A01, A00, BD, BA, B6, B3, CB, MD); \
		PERM_ELT(A02, A01, BE, BB, B7, B4, CA, ME); \
		PERM_ELT(A03, A02, BF, BC, B8, B5, C9, MF); \
	} while (0)

#define PERM_STEP_1   do { \
		PERM_ELT(A04, A03, B0, BD, B9, B6, C8, M0); \
		PERM_ELT(A05, A04, B1, BE, BA, B7, C7, M1); \
		PERM_ELT(A06, A05, B2, BF, BB, B8, C6, M2); \
		PERM_ELT(A07, A06, B3, B0, BC, B9, C5, M3); \
		PERM_ELT(A08, A07, B4, B1, BD, BA, C4, M4); \
		PERM_ELT(A09, A08, B5, B2, BE, BB, C3, M5); \
		PERM_ELT(A0A, A09, B6, B3, BF, BC, C2, M6); \
		PERM_ELT(A0B, A0A, B7, B4, B0, BD, C1, M7); \
		PERM_ELT(A00, A0B, B8, B5, B1, BE, C0, M8); \
		PERM_ELT(A01, A00, B9, B6, B2, BF, CF, M9); \
		PERM_ELT(A02, A01, BA, B7, B3, B0, CE, MA); \
		PERM_ELT(A03, A02, BB, B8, B4, B1, CD, MB); \
		PERM_ELT(A04, A03, BC, B9, B5, B2, CC, MC); \
		PERM_ELT(A05, A04, BD, BA, B6, B3, CB, MD); \
		PERM_ELT(A06, A05, BE, BB, B7, B4, CA, ME); \
		PERM_ELT(A07, A06, BF, BC, B8, B5, C9, MF); \
	} while (0)

#define PERM_STEP_2   do { \
		PERM_ELT(A08, A07, B0, BD, B9, B6, C8, M0); \
		PERM_ELT(A09, A08, B1, BE, BA, B7, C7, M1); \
		PERM_ELT(A0A, A09, B2, BF, BB, B8, C6, M2); \
		PERM_ELT(A0B, A0A, B3, B0, BC, B9, C5, M3); \
		PERM_ELT(A00, A0B, B4, B1, BD, BA, C4, M4); \
		PERM_ELT(A01, A00, B5, B2, BE, BB, C3, M5); \
		PERM_ELT(A02, A01, B6, B3, BF, BC, C2, M6); \
		PERM_ELT(A03, A02, B7, B4, B0, BD, C1, M7); \
		PERM_ELT(A04, A03, B8, B5, B1, BE, C0, M8); \
		PERM_ELT(A05, A04, B9, B6, B2, BF, CF, M9); \
		PERM_ELT(A06, A05, BA, B7, B3, B0, CE, MA); \
		PERM_ELT(A07, A06, BB, B8, B4, B1, CD, MB); \
		PERM_ELT(A08, A07, BC, B9, B5, B2, CC, MC); \
		PERM_ELT(A09, A08, BD, BA, B6, B3, CB, MD); \
		PERM_ELT(A0A, A09, BE, BB, B7, B4, CA, ME); \
		PERM_ELT(A0B, A0A, BF, BC, B8, B5, C9, MF); \
	} while (0)

#define APPLY_P   do { \
		B0 = T32(B0 << 17) | (B0 >> 15); \
		B1 = T32(B1 << 17) | (B1 >> 15); \
		B2 = T32(B2 << 17) | (B2 >> 15); \
		B3 = T32(B3 << 17) | (B3 >> 15); \
		B4 = T32(B4 << 17) | (B4 >> 15); \
		B5 = T32(B5 << 17) | (B5 >> 15); \
		B6 = T32(B6 << 17) | (B6 >> 15); \
		B7 = T32(B7 << 17) | (B7 >> 15); \
		B8 = T32(B8 << 17) | (B8 >> 15); \
		B9 = T32(B9 << 17) | (B9 >> 15); \
		BA = T32(BA << 17) | (BA >> 15); \
		BB = T32(BB << 17) | (BB >> 15); \
		BC = T32(BC << 17) | (BC >> 15); \
		BD = T32(BD << 17) | (BD >> 15); \
		BE = T32(BE << 17) | (BE >> 15); \
		BF = T32(BF << 17) | (BF >> 15); \
		PERM_STEP_0; \
		PERM_STEP_1; \
		PERM_STEP_2; \
		A0B = T32(A0B + C6); \
		A0A = T32(A0A + C5); \
		A09 = T32(A09 + C4); \
		A08 = T32(A08 + C3); \
		A07 = T32(A07 + C2); \
		A06 = T32(A06 + C1); \
		A05 = T32(A05 + C0); \
		A04 = T32(A04 + CF); \
		A03 = T32(A03 + CE); \
		A02 = T32(A02 + CD); \
		A01 = T32(A01 + CC); \
		A00 = T32(A00 + CB); \
		A0B = T32(A0B + CA); \
		A0A = T32(A0A + C9); \
		A09 = T32(A09 + C8); \
		A08 = T32(A08 + C7); \
		A07 = T32(A07 + C6); \
		A06 = T32(A06 + C5); \
		A05 = T32(A05 + C4); \
		A04 = T32(A04 + C3); \
		A03 = T32(A03 + C2); \
		A02 = T32(A02 + C1); \
		A01 = T32(A01 + C0); \
		A00 = T32(A00 + CF); \
		A0B = T32(A0B + CE); \
		A0A = T32(A0A + CD); \
		A09 = T32(A09 + CC); \
		A08 = T32(A08 + CB); \
		A07 = T32(A07 + CA); \
		A06 = T32(A06 + C9); \
		A05 = T32(A05 + C8); \
		A04 = T32(A04 + C7); \
		A03 = T32(A03 + C6); \
		A02 = T32(A02 + C5); \
		A01 = T32(A01 + C4); \
		A00 = T32(A00 + C3); \
	} while (0)

#define INCR_W   do { \
		if ((Wlow = T32(Wlow + 1)) == 0) \
			Whigh = T32(Whigh + 1); \
	} while (0)

__constant__
sph_u32 A_init_256[] = {
C32(0x52F84552), C32(0xE54B7999), C32(0x2D8EE3EC), C32(0xB9645191),
C32(0xE0078B86), C32(0xBB7C44C9), C32(0xD2B5C1CA), C32(0xB0D2EB8C),
C32(0x14CE5A45), C32(0x22AF50DC), C32(0xEFFDBC6B), C32(0xEB21B74A)
};

__constant__
sph_u32 B_init_256[] = {
C32(0xB555C6EE), C32(0x3E710596), C32(0xA72A652F), C32(0x9301515F),
C32(0xDA28C1FA), C32(0x696FD868), C32(0x9CB6BF72), C32(0x0AFE4002),
C32(0xA6E03615), C32(0x5138C1D4), C32(0xBE216306), C32(0xB38B8890),
C32(0x3EA8B96B), C32(0x3299ACE4), C32(0x30924DD4), C32(0x55CB34A5)
};

__constant__
sph_u32 C_init_256[] = {
C32(0xB405F031), C32(0xC4233EBA), C32(0xB3733979), C32(0xC0DD9D55),
C32(0xC51C28AE), C32(0xA327B8E1), C32(0x56C56167), C32(0xED614433),
C32(0x88B59D60), C32(0x60E2CEBA), C32(0x758B4B8B), C32(0x83E82A7F),
C32(0xBC968828), C32(0xE6E00BF7), C32(0xBA839E55), C32(0x9B491C60)
};

/*
* Part of this code was automatically generated (the part between
* the "BEGIN" and "END" markers).
*/

__device__
sph_u32 cuda_sph_dec32le_aligned(const void *src)
{
#if SPH_LITTLE_ENDIAN
	return *(const sph_u32 *)src;
#elif SPH_BIG_ENDIAN
#if SPH_SPARCV9_GCC && !SPH_NO_ASM
	sph_u32 tmp;

	__asm__ __volatile__ ("lda [%1]0x88,%0" : "=r" (tmp) : "r" (src));
	return tmp;
	/*
	* Not worth it generally.
	*
	#elif (SPH_PPC32_GCC || SPH_PPC64_GCC) && !SPH_NO_ASM
	sph_u32 tmp;

	__asm__ __volatile__ ("lwbrx %0,0,%1" : "=r" (tmp) : "r" (src));
	return tmp;
	*/
#else
	return sph_bswap32(*(const sph_u32 *)src);
#endif
#else
	return (sph_u32)(((const unsigned char *)src)[0])
		| ((sph_u32)(((const unsigned char *)src)[1]) << 8)
		| ((sph_u32)(((const unsigned char *)src)[2]) << 16)
		| ((sph_u32)(((const unsigned char *)src)[3]) << 24);
#endif
}

__device__
void cuda_shabal_init(void *cc, unsigned size)
{
	/*
	* We have precomputed initial states for all the supported
	* output bit lengths.
	*/
	sph_u32 *A_init, *B_init, *C_init;
	sph_shabal_context *sc;

	A_init = A_init_256;
	B_init = B_init_256;
	C_init = C_init_256;

	sc = (sph_shabal_context *)cc;
	
	memcpy(sc->A, A_init, sizeof sc->A);
	memcpy(sc->B, B_init, sizeof sc->B);
	memcpy(sc->C, C_init, sizeof sc->C);

	sc->Wlow = 1;
	sc->Whigh = 0;
	sc->ptr = 0;
}

__device__
void cuda_shabal_core(void *cc, const unsigned char *data, size_t len)
{
	sph_shabal_context *sc;
	unsigned char *buf;
	size_t ptr;
	DECL_STATE

	sc = (sph_shabal_context *)cc;
	buf = sc->buf;
	ptr = sc->ptr;

	/*
	* We do not want to copy the state to local variables if the
	* amount of data is less than what is needed to complete the
	* current block. Note that it is anyway suboptimal to call
	* this method many times for small chunks of data.
	*/
	if (len < (sizeof sc->buf) - ptr) {
		memcpy(buf + ptr, data, len);
		ptr += len;
		sc->ptr = ptr;
		return;
	}

	READ_STATE(sc);
	while (len > 0) {
		size_t clen;

		clen = (sizeof sc->buf) - ptr;
		if (clen > len)
			clen = len;
		memcpy(buf + ptr, data, clen);
		ptr += clen;
		data += clen;
		len -= clen;
		if (ptr == sizeof sc->buf) {
			DECODE_BLOCK;
			INPUT_BLOCK_ADD;
			XOR_W;
			APPLY_P;
			INPUT_BLOCK_SUB;
			SWAP_BC;
			INCR_W;
			ptr = 0;
		}
	}
	WRITE_STATE(sc);
	sc->ptr = ptr;
}

__device__
void cuda_sph_enc32le_aligned(void *dst, sph_u32 val)
{
#if SPH_LITTLE_ENDIAN
	*(sph_u32 *)dst = val;
#elif SPH_BIG_ENDIAN
	*(sph_u32 *)dst = sph_bswap32(val);
#else
	((unsigned char *)dst)[0] = val;
	((unsigned char *)dst)[1] = (val >> 8);
	((unsigned char *)dst)[2] = (val >> 16);
	((unsigned char *)dst)[3] = (val >> 24);
#endif
}

__device__
void cuda_shabal_close(void *cc, unsigned ub, unsigned n, void *dst)
{
	const auto size_word = 8;
	sph_shabal_context *sc;
	unsigned char *buf;
	size_t ptr;
	int i;
	unsigned z;
	union {
		unsigned char tmp_out[64];
		sph_u32 dummy;
	} u;
	size_t out_len;
	DECL_STATE

	sc = (sph_shabal_context *)cc;
	buf = sc->buf;
	ptr = sc->ptr;
	z = 0x80 >> n;
	buf[ptr] = ((ub & -z) | z) & 0xFF;
	memset(buf + ptr + 1, 0, (sizeof sc->buf) - (ptr + 1));
	READ_STATE(sc);
	DECODE_BLOCK;
	INPUT_BLOCK_ADD;
	XOR_W;
	APPLY_P;
	for (i = 0; i < 3; i ++) {
		SWAP_BC;
		XOR_W;
		APPLY_P;
	}

	/*
	* We just use our local variables; no need to go through
	* the state structure. In order to share some code, we
	* emit the relevant words into a temporary buffer, which
	* we finally copy into the destination array.
	*/
	cuda_sph_enc32le_aligned(u.tmp_out + 32, B8);
	cuda_sph_enc32le_aligned(u.tmp_out + 36, B9);
	cuda_sph_enc32le_aligned(u.tmp_out + 40, BA);
	cuda_sph_enc32le_aligned(u.tmp_out + 44, BB);
	cuda_sph_enc32le_aligned(u.tmp_out + 48, BC);
	cuda_sph_enc32le_aligned(u.tmp_out + 52, BD);
	cuda_sph_enc32le_aligned(u.tmp_out + 56, BE);
	cuda_sph_enc32le_aligned(u.tmp_out + 60, BF);

	out_len = size_word << 2;
	memcpy(dst, u.tmp_out + (sizeof u.tmp_out) - out_len, out_len);
	cuda_shabal_init(sc, size_word << 5);
}

__global__
void cuda_calculate_shabal(Burst::ScoopData* buffer, Poco::UInt64* deadlines, Poco::UInt64 len,
	const GensigData* gensig, Poco::UInt64 nonceStart, Poco::UInt64 baseTarget)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	sph_shabal256_context context;
	cuda_shabal_init(&context, 256);

	//printf("blockIdx.x = %i\n", blockIdx.x);
	//printf("blockIdx.x = %i, threadIdx.x = %i, index = %i\n", blockIdx.x, threadIdx.x, i);
	
	if (tid >= len)
	{
		//printf("blockIdx.x = %i, threadIdx.x = %i, index = %i\n", blockIdx.x, threadIdx.x, i);
		return;
	}

	Poco::UInt8 target[Settings::hashSize];
	auto test = buffer[tid];

	cuda_shabal_core(&context, (const unsigned char *)gensig, Settings::hashSize);
	cuda_shabal_core(&context, (const unsigned char *)&test, Settings::scoopSize);
	cuda_shabal_close(&context, 0, 0, &target[0]);

	Poco::UInt64 targetResult = 0;
	memcpy(&targetResult, &target[0], sizeof(Poco::UInt64));

	//auto buffer_deadline = (Poco::UInt64*)(buffer + tid);
	//*buffer_deadline = targetResult / baseTarget;
	deadlines[tid] = targetResult / baseTarget;
}

void cuda_calc_occupancy(int bufferSize, int& gridSize, int& blockSize)
{
	int minGridSize;

	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)cuda_calculate_shabal, 0, 0);

	gridSize = (bufferSize + blockSize - 1) / blockSize;
}

bool cuda_alloc_memory(Poco::UInt64 size, void** mem)
{
	if (size <= 0)
		return false;

	return hipMalloc((void**)&*mem, size) == hipSuccess;
}

bool cuda_copy_memory(Poco::UInt64 size, const void* from, void* to, MemoryCopyDirection copyDirection)
{
	if (size <= 0)
		return false;

	return hipMemcpy(to, from, size, copyDirection == MemoryCopyDirection::ToDevice ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost) == hipSuccess;
}

bool cuda_free_memory(void* mem)
{
	if (mem == nullptr)
		return false;

	return hipFree(mem) == hipSuccess;
}

bool cuda_calculate_shabal_host_preallocated(ScoopData* buffer, Poco::UInt64* deadlines, Poco::UInt64 bufferSize, const GensigData* gensig,
	Poco::UInt64 nonceStart, Poco::UInt64 baseTarget, std::string& errorString)
{
	auto blockSize = 0;
	auto minGridSize = 0;
	
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)cuda_calculate_shabal, 0, 0);

	int gridSize = (bufferSize + blockSize - 1) / blockSize;

	cuda_calculate_shabal<<<gridSize, blockSize>>>(buffer, deadlines, bufferSize, gensig, nonceStart, baseTarget);
	
	return !cuda_get_error(errorString);
}

bool cuda_reduce_best_deadline(Poco::UInt64* deadlines, size_t size, Poco::UInt64& minDeadline, Poco::UInt64& index, std::string& errorString)
{
	thrust::device_ptr<Poco::UInt64> deadlinesPtr(deadlines);
	auto pos = thrust::min_element(deadlinesPtr, deadlinesPtr + size);
	index = thrust::distance(deadlinesPtr, pos);

	if (!cuda_copy_memory(sizeof(Poco::UInt64), &deadlines[index], &minDeadline, MemoryCopyDirection::ToHost))
		return false;

	return !cuda_get_error(errorString);
}

bool cuda_get_error(std::string& errorString)
{
	const auto err = hipPeekAtLastError();

	if (err != hipSuccess)
	{
		errorString = hipGetErrorString(err);
		return true;
	}

	return false;
}

bool cuda_get_devices(std::vector<std::string>& devices)
{
	auto count = 0;

	if (hipGetDeviceCount(&count) != hipSuccess)
		return false;

	devices.clear();
	
	for (auto i = 0; i < count; ++i)
	{
		hipDeviceProp_t prop;
		
		if (hipGetDeviceProperties(&prop, i) != hipSuccess)
			continue;

		devices.emplace_back(prop.name);
	}

	return true;
}

bool cuda_set_device(unsigned index)
{
	return hipSetDevice(static_cast<int>(index)) == hipSuccess;
}
